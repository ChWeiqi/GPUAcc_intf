#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gpuacc.h"


__global__ void gpu_run_kernel(void *arg, gpu_run_fn fn) {
    fn(arg);
}

int gpu_run(void *arg, int arg_sz, gpu_run_fn fn) {
    if (fn == NULL) {
        fprintf(stderr, "Function pointer is NULL\n");
        return -1;
    }

    hipError_t err;

    // allocate memory for parameters
    void *d_arg;
    err = hipMalloc(&d_arg, arg_sz);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for parameters: %s\n", hipGetErrorString(err));
        return -1;
    }

    // copy parameters to device memory
    err = hipMemcpy(d_arg, arg, arg_sz, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy parameters to device memory: %s\n", hipGetErrorString(err));
        return -1;
    }

    // allocate memory for function
    gpu_run_fn d_fn;
    err = hipMalloc(&d_fn, sizeof(gpu_run_fn));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for function: %s\n", hipGetErrorString(err));
        return -1;
    }

    // copy function to device memory
    err = hipMemcpy((void*)d_fn, (void*)&fn, sizeof(gpu_run_fn), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy function to device memory: %s\n", hipGetErrorString(err));
        return -1;
    }

    gpu_run_kernel<<<1, 1>>>(d_arg, d_fn);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
        return -1;
    }

    // wait for kernel to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to synchronize device: %s\n", hipGetErrorString(err));
        return -1;
    }

    // free device memory
    hipFree(d_arg);
    hipFree((void*)d_fn);

    return 0;
}